﻿
#include "hip/hip_runtime.h"
#include ""
#include "antlr4-runtime.h"
#include "libs/qasm2Lexer.h"
#include "libs/qasm2Parser.h"
#include "libs/qasm2Visitor.h"
#include "libs/qasm2BaseVisitor.h"
#include "libs/staging.h"
#include "libs/CPUDevice.h"
#include "libs/GPUDevice.cuh"
#include "libs/Measurement.h"
#include "libs/JSONify.h"
#include <Windows.h>
#include <string>
#include <fstream>
#include <iostream>
#include <chrono>

#include <stdio.h>

#include "test/ValkyrieTests.h"

using namespace antlr4;

// getexepath allows vakyrie to resolve the directory it is operating in
std::string getexepath()
{
    char result[MAX_PATH];
    return std::string(result, GetModuleFileName(NULL, result, MAX_PATH));
}
// DisplayHeader is used during info print command to display GPU devices connected
void DisplayHeader();
// printHelp will print help if the user enters an invalid set of command line options
void printHelp();
// resolveDeviceType resolves what type of device the user wants to rpocess on
DeviceType resolveDeviceType(std::vector<std::string> arguments);
// fetchFileName finds the file name specified by the user
std::string fetchFileName(std::vector<std::string> arguments);
// CPURun performs CPU execution of the target QASM code
void CPURun(std::string filename, bool SV, bool jsonMode);
// GPURun performs GPU execution of the target QASM code
void GPURun(std::string filename, bool SV, bool jsonMode);
// handleInfoRequest prints the info requested by user in command line options
void handleInfoRequest(std::vector<std::string> arguments);
// resolveComputeMode resolves whether the user wants fast or statevector compute modes
bool resolveComputeMode(std::vector<std::string> arguments);
// resolveJSONPrint resolves whether this is a VisualQ call which requires json output
bool resolveJsonPrint(std::vector<std::string> arguments);

// timeCPUExecution is used for experimentation and metric gathering
void timeCPUExecution() {
    auto begin = std::chrono::high_resolution_clock::now();
    std::ifstream stream;    
    stream.open("output.qasm");
    ANTLRInputStream input(stream);

    qasm2Lexer lexer(&input);
    CommonTokenStream tokens(&lexer);
    qasm2Parser parser(&tokens);

    qasm2Parser::MainprogContext* tree = parser.mainprog();

    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);
    std::vector<Register> registers = visitor.getRegisters();
    std::vector<GateRequest> gateRequests = visitor.getGates();
    Stager stage = Stager();
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);
    CPUDevice device = CPUDevice();
    device.run(stage.getRegisters(), blocks);
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << std::endl;
    device.prettyPrintQubitStates(device.revealQuantumState());
}
// timeGPUExecution is used for experimentation and metric gathering
void timeGPUExecution() {
    auto begin = std::chrono::high_resolution_clock::now();
    std::ifstream stream;
    stream.open("output.qasm");
    ANTLRInputStream input(stream);

    qasm2Lexer lexer(&input);
    CommonTokenStream tokens(&lexer);
    qasm2Parser parser(&tokens);

    qasm2Parser::MainprogContext* tree = parser.mainprog();

    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);
    std::vector<Register> registers = visitor.getRegisters();
    std::vector<GateRequest> gateRequests = visitor.getGates();
    Stager stage = Stager();
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);
    GPUDevice deviceG = GPUDevice();
    deviceG.run(stage.getRegisters(), blocks);
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << std::endl;
    deviceG.prettyPrintQubitStates(deviceG.revealQuantumState());
    MeasurementCalculator measure = MeasurementCalculator(registers);
    measure.registerHandover(deviceG.revealQuantumState());
    measure.measureAll();
    std::map<std::string, std::vector<int>> measuredMap_ = measure.returnMeasurementMap();
    std::cout << "Measurement complete" << std::endl;
    std::vector<MeasureCommand> commands = visitor.getMeasureCommands();
    measure.loadMeasureCommands(commands);
    measure.passMeasurementsIntoClassicalRegisters();
    Register cReg = measure.fetchRegister("c");
    std::cout << "Commands processed" << std::endl;
}


// main is the entrypoint of the program
int main(int argc, char *argv[])
{
    std::vector<std::string> arguments;
    for (int i = 1; i < argc; i++) {
        arguments.push_back(argv[i]);           // collect command line arguments
    }
    handleInfoRequest(arguments);               // check if information was requested by user and print

    DeviceType type = resolveDeviceType(arguments);         // calculate whether the user wants to use the CPU or GPU
    if (type == INVALID) {
        std::cout << "Invalid or No execution mode provided, specify -c or -g" << std::endl;
        printHelp();
        return 1;
    }

    std::string fileName = fetchFileName(arguments);        // resolve the qasm file the user wants to process
    if (fileName == "INVALID") {
        std::cout << "File not specified, please use -o <filename> to indicate which file Valkyrie should process" << std::endl;
        printHelp();
        return 1;
    }
    bool svMode = resolveComputeMode(arguments);            // resolve whether the user wanted to user statevector or fast compute mode
    bool jsonMode = resolveJsonPrint(arguments);            // reolve whether the user wants a JSON print at the end or normal print
    if (type == CPU_) {                                     // depending on requested devicetype run on CPU or GPU
        CPURun(fileName, svMode, jsonMode);
    }
    else {
        GPURun(fileName, svMode, jsonMode);      
    }
    return 0;
}

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

    std::cout << "CUDA version:   v" << CUDART_VERSION << std::endl;

    int devCount;
    hipGetDeviceCount(&devCount);
    std::cout << "CUDA Devices: " << std::endl << std::endl;

    for (int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
        std::cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
        std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
        std::cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
        std::cout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

        std::cout << "  Warp size:         " << props.warpSize << std::endl;
        std::cout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
        std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << std::endl;
        std::cout << std::endl;
    }
}

void printHelp() {
    std::cout << "Welcome to Valkyrie Help" << std::endl;
    std::cout << "Command line options" << std::endl;
    std::cout << "CPU execution mode: \t \t \t -c" << std::endl;
    std::cout << "GPU execution mode: \t \t \t -g" << std::endl;
    std::cout << "Path to file: \t \t \t \t -o <filepath>" << std::endl;
    std::cout << "State vector computation: -sv" << std::endl;
}

DeviceType resolveDeviceType(std::vector<std::string> arguments) {
    DeviceType val = INVALID;
    for (std::string argument : arguments) {
        if (argument == "-g") {
            val = GPU_;
            break;
        }
        if (argument == "-c") {
            val = CPU_;
            break;
        }
    }
    return val;
}

bool resolveComputeMode(std::vector<std::string> arguments) {
    for (std::string argument : arguments) {
        if (argument == "-sv") {
            return true;
        }
    }
    return false;
}

bool resolveJsonPrint(std::vector<std::string> arguments) {
    for (std::string argument : arguments) {
        if (argument == "-json") {
            return true;
        }
    }
    return false;
}

std::string fetchFileName(std::vector<std::string> arguments) {
    std::string returnVal = "INVALID";
    if (arguments.size() == 0) {
        return returnVal;
    }
    for (int i = 0; i < arguments.size()-1; i++) {
        if (arguments[i] == "-o") {
            return arguments[i + 1];
        }
    }
    return returnVal;
}

void CPURun(std::string filename, bool SV, bool jsonMode) {
    std::ifstream stream;
    stream.open(filename);          // Open File requested
    if (!stream.is_open()) {
        std::cout << "Couldn't find file specified" << std::endl;
        printHelp();
        return;
    }
    ANTLRInputStream input(stream);             // Convert filestream to ANTLR stream

    qasm2Lexer lexer(&input);                   // Lex file
    CommonTokenStream tokens(&lexer);           // get the tokens
    qasm2Parser parser(&tokens);                // send to antlr parser

    qasm2Parser::MainprogContext* tree = parser.mainprog();             // Fetch AST tree

    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);                                        // Use custom visitor to process information
    std::vector<Register> registers = visitor.getRegisters();           // Get registers defined by user
    std::vector<GateRequest> gateRequests = visitor.getGates();         // Get gates defined by user
    Stager stage = Stager();                                            
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);          // User stager to convert parsed information into calculation commands
    CPUDevice device = CPUDevice();
    if (!SV) {                                                          // If we are in statevector compute mode, run in statevector mode
        device.run(stage.getRegisters(), blocks);
    }
    else {
        device.runSV(stage.getRegisters(), blocks);
    }    
    StateVectorMeasurement measure = StateVectorMeasurement(device.getStateVector(), registers);        // Initialise statevector measurement
    measure.measure();
    std::vector<MeasureCommand> commands = visitor.getMeasureCommands();
    measure.loadMeasureCommands(commands);
    measure.passMeasurementsIntoClassicalRegisters();
    if (!jsonMode) {
        measure.printClassicalRegisters();
    }
    else {
        JSONify json = JSONify(measure.getAllRegisters(), device.getStateVector());                     // If requested print in JSON format
        json.printJSON();
    }
}

void GPURun(std::string filename, bool SV, bool jsonMode) {
    std::ifstream stream;
    stream.open(filename);          // Open File requested
    if (!stream.is_open()) {
        std::cout << "Couldn't find file specified" << std::endl;
        printHelp();
        return;
    }
    ANTLRInputStream input(stream);                 // Convert filestream to ANTLR stream

    qasm2Lexer lexer(&input);                       // Lex file
    CommonTokenStream tokens(&lexer);               // get the tokens
    qasm2Parser parser(&tokens);                    // send to antlr parser

    qasm2Parser::MainprogContext* tree = parser.mainprog();             // Fetch AST tree

    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);                                        // Use custom visitor to process information
    std::vector<Register> registers = visitor.getRegisters();           // Get registers defined by user
    std::vector<GateRequest> gateRequests = visitor.getGates();         // Get gates defined by user
    Stager stage = Stager();
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);          // User stager to convert parsed information into calculation commands
    GPUDevice device = GPUDevice();
    if (!SV) {                                                          // If we are in statevector compute mode, run in statevector mode
        device.run(stage.getRegisters(), blocks);
    }
    else {
        device.runSV(stage.getRegisters(), blocks);
    }
    StateVectorMeasurement measure = StateVectorMeasurement(device.getStateVector(), registers);        // Initialise statevector measurement
    measure.measure();
    std::vector<MeasureCommand> commands = visitor.getMeasureCommands();
    measure.loadMeasureCommands(commands);
    measure.passMeasurementsIntoClassicalRegisters();
    if (!jsonMode) {
        measure.printClassicalRegisters();
    }
    else {
        JSONify json = JSONify(measure.getAllRegisters(), device.getStateVector());                     // If requested print in JSON format
        json.printJSON();
    }
}

void handleInfoRequest(std::vector<std::string> arguments)
{
    for (auto argument : arguments) {
        if (argument == "-gpuInfo") {
            DisplayHeader();
        }
        if (argument == "-test") {
            ValkyrieTests tester = ValkyrieTests();
            tester.runTests();
            std::cout << "Number of Tests passed: " << tester.noPassed() << std::endl;
            std::cout << "Test pass percentage: " << tester.getPercentagePassed() << std::endl;
            if (tester.getPercentagePassed() != 100.0) {
                for (auto fail : tester.testsFailed()) {
                    std::cout << "Test Failed: " << fail << std::endl;
                }
            }
        }
    }
}
