#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "antlr4-runtime.h"
#include "libs/qasm2Lexer.h"
#include "libs/qasm2Parser.h"
#include "libs/qasm2Visitor.h"
#include "libs/qasm2BaseVisitor.h"
#include "libs/staging.h"
#include "libs/CPUDevice.h"
#include "libs/GPUDevice.cuh"
#include "libs/Measurement.h"
#include "libs/JSONify.h"
#include <Windows.h>
#include <string>
#include <fstream>
#include <iostream>
#include <chrono>

#include <stdio.h>

#include "test/ValkyrieTests.h"

using namespace antlr4;

// getexepath allows vakyrie to resolve the directory it is operating in
std::string getexepath()
{
    char result[MAX_PATH];
    return std::string(result, GetModuleFileName(NULL, result, MAX_PATH));
}
// DisplayHeader is used during info print command to display GPU devices connected
void DisplayHeader();
// printHelp will print help if the user enters an invalid set of command line options
void printHelp();
// resolveDeviceType resolves what type of device the user wants to rpocess on
DeviceType resolveDeviceType(std::vector<std::string> arguments);
// fetchFileName finds the file name specified by the user
std::string fetchFileName(std::vector<std::string> arguments);
// CPURun performs CPU execution of the target QASM code
void CPURun(std::string filename, bool SV, bool jsonMode);
// GPURun performs GPU execution of the target QASM code
void GPURun(std::string filename, bool SV, bool jsonMode);
// handleInfoRequest prints the info requested by user in command line options
void handleInfoRequest(std::vector<std::string> arguments);
// resolveComputeMode resolves whether the user wants fast or statevector compute modes
bool resolveComputeMode(std::vector<std::string> arguments);
// resolveJSONPrint resolves whether this is a VisualQ call which requires json output
bool resolveJsonPrint(std::vector<std::string> arguments);

enum timingPoint
{
    NONE_,
    FULL,
    PARSE,
    STAGE,
    EXECUTION
};

// resolveTimingRequest
timingPoint resolveTimingRequest(std::vector<std::string> arguments);

// timeCPUExecution is used for experimentation and metric gathering
void timeCPUExecution(std::string filename, bool SV, bool jsonMode, timingPoint point) {
    // Start clock
    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;
    if (point == FULL || point == PARSE) {
        begin = std::chrono::high_resolution_clock::now();
    }
    std::ifstream stream;
    stream.open(filename);          // Open File requested
    if (!stream.is_open()) {
        std::cout << "Couldn't find file specified" << std::endl;
        printHelp();
        return;
    }
    ANTLRInputStream input(stream);                 // Convert filestream to ANTLR stream
    qasm2Lexer lexer(&input);                       // Lex file
    CommonTokenStream tokens(&lexer);               // get the tokens
    qasm2Parser parser(&tokens);                    // send to antlr parser
    qasm2Parser::MainprogContext* tree = parser.mainprog();             // Fetch AST tree
    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);                                        // Use custom visitor to process information
    std::vector<Register> registers = visitor.getRegisters();           // Get registers defined by user
    std::vector<GateRequest> gateRequests = visitor.getGates();         // Get gates defined by user   
    if (point == PARSE) {
        end = std::chrono::high_resolution_clock::now();
    }
    if (point == STAGE) {
        begin = std::chrono::high_resolution_clock::now();
    }
    Stager stage = Stager();
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);          // User stager to convert parsed information into calculation commands       
    if (point == STAGE) {
        end = std::chrono::high_resolution_clock::now();
    }
    if (point == EXECUTION) {
        begin = std::chrono::high_resolution_clock::now();
    }
    CPUDevice device = CPUDevice();
    if (!SV) {                                                          // If we are in statevector compute mode, run in statevector mode
        device.run(stage.getRegisters(), blocks);
    }
    else {
        device.runSV(stage.getRegisters(), blocks);
    }
    if (point == EXECUTION) {
        end = std::chrono::high_resolution_clock::now();
    }
    StateVectorMeasurement measure = StateVectorMeasurement(device.getStateVector(), registers);        // Initialise statevector measurement
    measure.measure();
    if (point == FULL) {
        end = std::chrono::high_resolution_clock::now();
    }
    std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << std::endl;   
}
// timeGPUExecution is used for experimentation and metric gathering
void timeGPUExecution(std::string filename, bool SV, bool jsonMode, timingPoint point) {
    // Start clock
    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;
    if (point == FULL || point == PARSE) {
        begin = std::chrono::high_resolution_clock::now();
    }
    std::ifstream stream;
    stream.open(filename);          // Open File requested
    if (!stream.is_open()) {
        std::cout << "Couldn't find file specified" << std::endl;
        printHelp();
        return;
    }
    ANTLRInputStream input(stream);                 // Convert filestream to ANTLR stream
    qasm2Lexer lexer(&input);                       // Lex file
    CommonTokenStream tokens(&lexer);               // get the tokens
    qasm2Parser parser(&tokens);                    // send to antlr parser
    qasm2Parser::MainprogContext* tree = parser.mainprog();             // Fetch AST tree
    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);                                        // Use custom visitor to process information
    std::vector<Register> registers = visitor.getRegisters();           // Get registers defined by user
    std::vector<GateRequest> gateRequests = visitor.getGates();         // Get gates defined by user   
    if (point == PARSE) {
        end = std::chrono::high_resolution_clock::now();
    }
    if (point == STAGE) {
        begin = std::chrono::high_resolution_clock::now();
    }
    Stager stage = Stager();
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);          // User stager to convert parsed information into calculation commands       
    if (point == STAGE) {
        end = std::chrono::high_resolution_clock::now();
    }
    if (point == EXECUTION) {
        begin = std::chrono::high_resolution_clock::now();
    }
    GPUDevice device = GPUDevice();
    if (!SV) {                                                          // If we are in statevector compute mode, run in statevector mode
        device.run(stage.getRegisters(), blocks);
    }
    else {
        device.runSV(stage.getRegisters(), blocks);
    }
    if (point == EXECUTION) {
        end = std::chrono::high_resolution_clock::now();
    }
    StateVectorMeasurement measure = StateVectorMeasurement(device.getStateVector(), registers);        // Initialise statevector measurement
    measure.measure();
    if (point == FULL) {
        end = std::chrono::high_resolution_clock::now();
    }
    std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << std::endl;
}


// main is the entrypoint of the program
int main(int argc, char *argv[])
{
    std::vector<std::string> arguments;
    for (int i = 1; i < argc; i++) {
        arguments.push_back(argv[i]);           // collect command line arguments
    }
    handleInfoRequest(arguments);               // check if information was requested by user and print

    DeviceType type = resolveDeviceType(arguments);         // calculate whether the user wants to use the CPU or GPU
    if (type == INVALID) {
        std::cout << "Invalid or No execution mode provided, specify -c or -g" << std::endl;
        printHelp();
        return 1;
    }

    std::string fileName = fetchFileName(arguments);        // resolve the qasm file the user wants to process
    if (fileName == "INVALID") {
        std::cout << "File not specified, please use -o <filename> to indicate which file Valkyrie should process" << std::endl;
        printHelp();
        return 1;
    }
    bool svMode = resolveComputeMode(arguments);            // resolve whether the user wanted to user statevector or fast compute mode
    bool jsonMode = resolveJsonPrint(arguments);            // reolve whether the user wants a JSON print at the end or normal print
    timingPoint timing = resolveTimingRequest(arguments);
    if (timing != NONE_) {
        if (type == CPU_) {                                     // depending on requested devicetype run on CPU or GPU
            for (int i = 0; i < 121; i++) {
                timeCPUExecution(fileName, svMode, jsonMode, timing);
            }
        }
        else {
            for (int i = 0; i < 121; i++) {
                timeGPUExecution(fileName, svMode, jsonMode, timing);
            }
        }
        return 0;
    }
    if (type == CPU_) {                                     // depending on requested devicetype run on CPU or GPU
        CPURun(fileName, svMode, jsonMode);
    }
    else {
        GPURun(fileName, svMode, jsonMode);      
    }
    return 0;
}

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

    std::cout << "CUDA version:   v" << CUDART_VERSION << std::endl;

    int devCount;
    hipGetDeviceCount(&devCount);
    std::cout << "CUDA Devices: " << std::endl << std::endl;

    for (int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
        std::cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
        std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
        std::cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
        std::cout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

        std::cout << "  Warp size:         " << props.warpSize << std::endl;
        std::cout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
        std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << std::endl;
        std::cout << std::endl;
    }
}

void printHelp() {
    std::cout << "Welcome to Valkyrie Help" << std::endl;
    std::cout << "Command line options" << std::endl;
    std::cout << "CPU execution mode: \t \t \t -c" << std::endl;
    std::cout << "GPU execution mode: \t \t \t -g" << std::endl;
    std::cout << "Path to file: \t \t \t \t -o <filepath>" << std::endl;
    std::cout << "State vector computation: -sv" << std::endl;
}

DeviceType resolveDeviceType(std::vector<std::string> arguments) {
    DeviceType val = INVALID;
    for (std::string argument : arguments) {
        if (argument == "-g") {
            val = GPU_;
            break;
        }
        if (argument == "-c") {
            val = CPU_;
            break;
        }
    }
    return val;
}

bool resolveComputeMode(std::vector<std::string> arguments) {
    for (std::string argument : arguments) {
        if (argument == "-sv") {
            return true;
        }
    }
    return false;
}

bool resolveJsonPrint(std::vector<std::string> arguments) {
    for (std::string argument : arguments) {
        if (argument == "-json") {
            return true;
        }
    }
    return false;
}

timingPoint resolveTimingRequest(std::vector<std::string> arguments) {
    for (int i = 0; i < arguments.size(); i++) {
        if (arguments[i] == "-time") {
            if (i != arguments.size() - 1) {
                if (arguments[i + 1] == "parse") {
                    return PARSE;
                }
                if (arguments[i + 1] == "staging") {
                    return STAGE;
                }
                if (arguments[i + 1] == "execution") {
                    return EXECUTION;
                }
            }
            return FULL;
        }
    }
    return NONE_;
}

std::string fetchFileName(std::vector<std::string> arguments) {
    std::string returnVal = "INVALID";
    if (arguments.size() == 0) {
        return returnVal;
    }
    for (int i = 0; i < arguments.size()-1; i++) {
        if (arguments[i] == "-o") {
            return arguments[i + 1];
        }
    }
    return returnVal;
}

void CPURun(std::string filename, bool SV, bool jsonMode) {
    std::ifstream stream;
    stream.open(filename);          // Open File requested
    if (!stream.is_open()) {
        std::cout << "Couldn't find file specified" << std::endl;
        printHelp();
        return;
    }
    ANTLRInputStream input(stream);             // Convert filestream to ANTLR stream

    qasm2Lexer lexer(&input);                   // Lex file
    CommonTokenStream tokens(&lexer);           // get the tokens
    qasm2Parser parser(&tokens);                // send to antlr parser

    qasm2Parser::MainprogContext* tree = parser.mainprog();             // Fetch AST tree

    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);                                        // Use custom visitor to process information
    std::vector<Register> registers = visitor.getRegisters();           // Get registers defined by user
    std::vector<GateRequest> gateRequests = visitor.getGates();         // Get gates defined by user
    Stager stage = Stager();                                            
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);          // User stager to convert parsed information into calculation commands
    CPUDevice device = CPUDevice();
    if (!SV) {                                                          // If we are in statevector compute mode, run in statevector mode
        device.run(stage.getRegisters(), blocks);
    }
    else {
        device.runSV(stage.getRegisters(), blocks);
    }    
    StateVectorMeasurement measure = StateVectorMeasurement(device.getStateVector(), registers);        // Initialise statevector measurement
    measure.measure();
    std::vector<MeasureCommand> commands = visitor.getMeasureCommands();
    measure.loadMeasureCommands(commands);
    measure.passMeasurementsIntoClassicalRegisters();
    if (!jsonMode) {
        measure.printClassicalRegisters();
    }
    else {
        JSONify json = JSONify(measure.getAllRegisters(), device.getStateVector());                     // If requested print in JSON format
        json.printJSON();
    }
}

void GPURun(std::string filename, bool SV, bool jsonMode) {
    std::ifstream stream;
    stream.open(filename);          // Open File requested
    if (!stream.is_open()) {
        std::cout << "Couldn't find file specified" << std::endl;
        printHelp();
        return;
    }
    ANTLRInputStream input(stream);                 // Convert filestream to ANTLR stream

    qasm2Lexer lexer(&input);                       // Lex file
    CommonTokenStream tokens(&lexer);               // get the tokens
    qasm2Parser parser(&tokens);                    // send to antlr parser

    qasm2Parser::MainprogContext* tree = parser.mainprog();             // Fetch AST tree

    qasm2BaseVisitor visitor;
    visitor.visitMainprog(tree);                                        // Use custom visitor to process information
    std::vector<Register> registers = visitor.getRegisters();           // Get registers defined by user
    std::vector<GateRequest> gateRequests = visitor.getGates();         // Get gates defined by user
    Stager stage = Stager();
    std::vector<ConcurrentBlock> blocks = stage.stageInformation(registers, gateRequests);          // User stager to convert parsed information into calculation commands
    GPUDevice device = GPUDevice();
    if (!SV) {                                                          // If we are in statevector compute mode, run in statevector mode
        device.run(stage.getRegisters(), blocks);
    }
    else {
        device.runSV(stage.getRegisters(), blocks);
    }
    StateVectorMeasurement measure = StateVectorMeasurement(device.getStateVector(), registers);        // Initialise statevector measurement
    measure.measure();
    std::vector<MeasureCommand> commands = visitor.getMeasureCommands();
    measure.loadMeasureCommands(commands);
    measure.passMeasurementsIntoClassicalRegisters();
    if (!jsonMode) {
        measure.printClassicalRegisters();
    }
    else {
        JSONify json = JSONify(measure.getAllRegisters(), device.getStateVector());                     // If requested print in JSON format
        json.printJSON();
    }
}

void handleInfoRequest(std::vector<std::string> arguments)
{
    for (auto argument : arguments) {
        if (argument == "-gpuInfo") {
            DisplayHeader();
        }
        if (argument == "-test") {
            ValkyrieTests tester = ValkyrieTests();
            tester.runTests();
            std::cout << "Number of Tests passed: " << tester.noPassed() << std::endl;
            std::cout << "Test pass percentage: " << tester.getPercentagePassed() << std::endl;
            if (tester.getPercentagePassed() != 100.0) {
                for (auto fail : tester.testsFailed()) {
                    std::cout << "Test Failed: " << fail << std::endl;
                }
            }
        }
    }
}
