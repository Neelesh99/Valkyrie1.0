#include "hip/hip_runtime.h"
﻿#pragma once
#include "GPUDevice.cuh"
#include "hip/hip_complex.h"
#include <cmath>
#include <stdio.h>
#include "GPUCompute.cuh"
#include "GateUtilitiesGPU.cuh"

using namespace std::complex_literals;
const double ROOT2INV = 1.0 / std::pow(2, 0.5);



std::vector<std::vector<std::complex<double>>> getGateMatrixGPU(GateRequest gate) {
	GateRequestType gateType = gate.getGateType();
	switch (gateType) {
	case I:
		return std::vector<std::vector<std::complex<double>>> { {1, 0}, { 0, 1 } };
		break;
	case h:
		return std::vector<std::vector<std::complex<double>>> { {ROOT2INV, ROOT2INV}, { ROOT2INV, -1.0 * ROOT2INV } };
		break;
	case cx:
		return std::vector<std::vector<std::complex<double>>> { {1, 0, 0, 0}, { 0, 1, 0, 0 }, { 0, 0, 0, 1 }, { 0, 0, 1, 0 } };
		break;
	case U:
		return buildU3GateGPU(gate);
		break;
	case CX:
		return std::vector<std::vector<std::complex<double>>> { {1, 0, 0, 0}, { 0, 1, 0, 0 }, { 0, 0, 0, 1 }, { 0, 0, 1, 0 } };
		break;
	}
}


Qubit* GPUQubitFactory::generateQubit()
{
	// Allocate heap memory for Qubit values
	std::complex<double>* s0 = new std::complex<double>;
	std::complex<double>* s1 = new std::complex<double>;
	*s0 = 1.0;
	*s1 = 0.0;
	// Allocate heap memory for Qubit and store values
	Qubit* generatedQubit = new Qubit(s0, s1);
	// Push into qubit tracker for deletion
	qubits_.push_back(generatedQubit);

	return generatedQubit;
}

GPUQubitFactory::~GPUQubitFactory()
{
	for (auto qubit : qubits_) {
		delete qubit->fetch(0);
		delete qubit->fetch(1);
		delete qubit;
	}
}

Gate* GPUGateFactory::generateGate(GateRequest request)
{
	std::vector<std::vector<std::complex<double>>> gateMatrix = getGateMatrixGPU(request);
	int gateM = gateMatrix.size();
	int gateN = gateMatrix[0].size();

	Gate* generatedGate = new Gate(gateM, gateN, gateMatrix);
	gates_.push_back(generatedGate);
	return generatedGate;
}

GPUGateFactory::~GPUGateFactory()
{
	for (auto gate : gates_) {
		delete gate;
	}
}

std::vector<SVPair> GPUQuantumCircuit::zipSVPairs(std::vector<std::string> names, std::vector<int> locs)
{
	std::vector<SVPair> values;
	for (int i = 0; i < names.size(); i++) {
		values.push_back(SVPair(names[i], locs[i]));
	}
	return values;
}

void GPUQuantumCircuit::loadQubitMap(std::map<std::string, std::vector<Qubit*>> qubitMap)
{
	qubitMap_ = qubitMap;
	sv_ = new StateVector(&qubitMap_);
	sv_->tensorProduct();
}

void GPUQuantumCircuit::loadBlock(ConcurrentBlock block)
{
	std::vector<GateRequest> gates = block.getGates();
	std::vector<Calculation> calcs;
	for (auto gate : gates) {
		std::vector<std::string> registers = gate.getRegisters();
		std::vector<int> locations = gate.getLocations();
		std::vector<Qubit*> qubitValues;
		for (int i = 0; i < registers.size(); i++) {
			qubitValues.push_back(qubitMap_[registers[i]][locations[i]]);
		}
		Gate* gateTrue = gateFactory_->generateGate(gate);
		std::vector<SVPair> svPairs = zipSVPairs(registers, locations);
		Calculation calc = Calculation(gateTrue, qubitValues, svPairs);
		calcs.push_back(calc);
	}
	calculations_.push_back(calcs);
}

std::vector<Calculation> GPUQuantumCircuit::getNextCalculation()
{
	if (calcCounter == calculations_.size() - 1) {
		done_ = true;
		return calculations_[calcCounter];
	}
	else {
		std::vector<Calculation> val = calculations_[calcCounter];
		calcCounter++;
		return val;
	}
}

std::map<std::string, std::vector<Qubit*>> GPUQuantumCircuit::returnResults()
{
	return qubitMap_;
}

StateVector* GPUQuantumCircuit::getStateVector()
{
	return sv_;
}

bool GPUQuantumCircuit::checkComplete()
{
	if (calculations_.size() == 0) {
		return true;
	}
	return done_;
}

std::vector<std::vector<std::complex<double>>> GPUQuantumProcessor::getCXResult(int n)
{
	// n is the number of qubits, we have to have n-2 I gates and then a CX gate at the end
	if (n < 2) {
		return std::vector<std::vector<std::complex<double>>>();
	}
	std::vector<std::vector<std::complex<double>>> output;
	// overall sidelength of resultant gate
	int dimOverall = std::pow(2, n);
	// number of I multiplications required
	int leftOver = n - 2;
	if (leftOver == 0) {
		output = { {1, 0, 0, 0}, {0, 1, 0, 0}, {0, 0, 0, 1}, {0, 0, 1, 0} };
		return output;
	}
	output.resize(dimOverall);
	for (int i = 0; i < dimOverall; i++) {
		std::vector<std::complex<double>> subVec;
		subVec.resize(dimOverall);
		output[i] = subVec;
	}
	for (int i = 0; i < std::pow(2, leftOver); i++) {
		output[4 * i][4 * i] = 1;
		output[4 * i + 1][4 * i + 1] = 1;
		output[4 * i + 2][4 * i + 3] = 1;
		output[4 * i + 3][4 * i + 2] = 1;
	}
	return output;
}

std::vector<std::vector<std::complex<double>>> GPUQuantumProcessor::getGenericUResult(Gate* gate, int n)
{
	// n is the number of qubits, we have to have n-2 I gates and then a CX gate at the end
	if (n < 1) {
		return std::vector<std::vector<std::complex<double>>>();
	}
	std::vector<std::vector<std::complex<double>>> output;
	// overall sidelength of resultant gate
	int dimOverall = std::pow(2, n);
	// number of I multiplications required
	int leftOver = n - 1;
	if (leftOver == 0) {
		output = gate->getArray();
		return output;
	}
	output.resize(dimOverall);
	for (int i = 0; i < dimOverall; i++) {
		std::vector<std::complex<double>> subVec;
		subVec.resize(dimOverall);
		output[i] = subVec;
	}
	for (int i = 0; i < std::pow(2, leftOver); i++) {
		output[2 * i][2 * i] = gate->fetchValue(0, 0);
		output[2 * i][2 * i + 1] = gate->fetchValue(0, 1);
		output[2 * i + 1][2 * i] = gate->fetchValue(1, 0);
		output[2 * i + 1][2 * i + 1] = gate->fetchValue(1, 1);
	}
	return output;
}

void GPUQuantumProcessor::loadCircuit(AbstractQuantumCircuit* circuit)
{
	circuit_ = circuit;
}

void GPUQuantumProcessor::calculate()
{
	// Generate initial arrays
	//hipDoubleComplex* initialValues;
	hipDoubleComplex* beforeGate;
	hipDoubleComplex* gateValues;
	hipDoubleComplex* afterGate;
	while (!circuit_->checkComplete()) {
		std::vector<Calculation> calcBlock = circuit_->getNextCalculation();
		for (auto calc : calcBlock) {	// parallelisation next iteration			
			Gate* gate = calc.getGate();
			int m = gate->getM();
			int n = gate->getN();
			int qubitN = m / 2;
			hipError_t cudaStatus;			
			// Allocate shared space
			cudaStatus = hipMalloc((void**)&beforeGate, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&afterGate, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&gateValues, (m*n) * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			std::vector<std::complex<double>> res = ValkGPULib::calculateGPU(beforeGate, gateValues, afterGate, calc.getGate(), calc.getQubits());
			if (res.size() == 2) {
				circuit_->getStateVector()->quickRefresh();
			}
			if (res.size() == 4) {
				circuit_->getStateVector()->modifyState(res, calc.getLocations()[0], calc.getLocations()[1]);
			}
			hipFree(beforeGate);
			hipFree(afterGate);
			hipFree(gateValues);
		}
	}
	return;
Error:
	hipFree(beforeGate);
	hipFree(afterGate);
	hipFree(gateValues);
}

void GPUQuantumProcessor::calculateWithStateVector()
{
	// Generate initial arrays
	//hipDoubleComplex* initialValues;
	hipDoubleComplex* beforeGate;
	hipDoubleComplex* gateValues;
	hipDoubleComplex* afterGate;
	while (!circuit_->checkComplete()) {
		std::vector<Calculation> calcBlock = circuit_->getNextCalculation();
		for (auto calc : calcBlock) {			
			Gate* gate = calc.getGate();
			int m = gate->getM();
			int n = gate->getN();
			int qubitN = m / 2;
			StateVector* sv = circuit_->getStateVector();
			int gateDim = sv->getState().size();
			std::vector<SVPair> newOrder = calc.getNewOrder(sv->getOrder());
			StateVector* reordered = sv->reorder(newOrder);
			std::vector<std::vector<std::complex<double>>> gateValuesV;
			if (m == 2) {
				gateValuesV = getGenericUResult(gate, sv->getN());
			}
			if (m == 4) {
				gateValuesV = getCXResult(sv->getN());
			}
			if (gateValuesV.size() == 0) {
				return;
			}
			hipError_t cudaStatus;
			// Allocate shared space
			cudaStatus = hipMalloc((void**)&beforeGate, gateDim * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&afterGate, gateDim * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&gateValues, (gateDim * gateDim) * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			std::vector<std::complex<double>> res;
			if (gateDim < 256) {	
				res = ValkGPULib::calculateGPUSV(beforeGate, gateValues, afterGate, reordered, gateValuesV);
			}
			else {
				// Ultra parallel
				res = ValkGPULib::calculateGPULargeSV(beforeGate, gateValues, afterGate, reordered, gateValuesV);
			}
			reordered->directModify(res);
			sv->reconcile(reordered);
			hipFree(beforeGate);
			hipFree(afterGate);
			hipFree(gateValues);
		}
	}
	return;
Error:
	hipFree(beforeGate);
	hipFree(afterGate);
	hipFree(gateValues);
}

std::map<std::string, std::vector<Qubit*>> GPUQuantumProcessor::qubitMapfetchQubitValues()
{
	return circuit_->returnResults();
}

void GPUDevice::loadRegister(Register registerx)
{
	if (registerx.isQuantum()) {
		QuantumRegister qReg = registerx.getQuantumRegister();
		std::string regName = qReg.getIdentifier();
		int width = qReg.getWidth();
		std::vector<Qubit*> registerQubits;
		for (int i = 0; i < width; i++) {
			registerQubits.push_back(qubitFactory->generateQubit());
		}
		registerMap.insert(std::pair<std::string, std::vector<Qubit*>>(regName, registerQubits));
	}
}

void GPUDevice::transferQubitMap()
{
	quantumCircuit->loadQubitMap(registerMap);
}

void GPUDevice::loadConcurrentBlock(ConcurrentBlock block)
{
	quantumCircuit->loadBlock(block);
}

void GPUDevice::runSimulation()
{
	quantumProcessor->loadCircuit(quantumCircuit);
	quantumProcessor->calculate();
}

void GPUDevice::runSimulationSV()
{
	quantumProcessor->loadCircuit(quantumCircuit);
	quantumProcessor->calculateWithStateVector();
}

void GPUDevice::run(std::vector<Register> registers, std::vector<ConcurrentBlock> blocks)
{
	for (auto reg : registers) {
		loadRegister(reg);
	}
	transferQubitMap();
	for (auto block : blocks) {
		loadConcurrentBlock(block);
	}
	runSimulation();
}

void GPUDevice::runSV(std::vector<Register> registers, std::vector<ConcurrentBlock> blocks)
{
	for (auto reg : registers) {
		loadRegister(reg);
	}
	transferQubitMap();
	for (auto block : blocks) {
		loadConcurrentBlock(block);
	}
	runSimulationSV();
}

std::map<std::string, std::vector<Qubit*>> GPUDevice::revealQuantumState()
{
	return quantumProcessor->qubitMapfetchQubitValues();
}
