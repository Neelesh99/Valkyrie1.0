#include "hip/hip_runtime.h"
﻿#include "GPUDevice.cuh"
#include "hip/hip_complex.h"
#include <cmath>
#include <stdio.h>
#include "GPUCompute.cuh"

using namespace std::complex_literals;
const double ROOT2INV = 1.0 / std::pow(2, 0.5);

std::vector<std::vector<std::complex<double>>> getGateMatrixGPU(GateRequestType gateType) {
	switch (gateType) {
	case I:
		return std::vector<std::vector<std::complex<double>>> { {1, 0}, { 0, 1 } };
		break;
	case h:
		return std::vector<std::vector<std::complex<double>>> { {ROOT2INV, ROOT2INV}, { ROOT2INV, -1.0 * ROOT2INV } };
		break;
	case cx:
		return std::vector<std::vector<std::complex<double>>> { {1, 0, 0, 0}, { 0, 1, 0, 0 }, { 0, 0, 0, 1 }, { 0, 0, 1, 0 } };
		break;
	}
}

hipDoubleComplex convertQubitComplex(std::complex<double> input) {
	return make_hipDoubleComplex(input.real(), input.imag());
}

std::complex<double> convertComplexQubit(hipDoubleComplex input) {
	return std::complex<double>(input.x, input.y);
}

hipDoubleComplex tensorProduct(std::vector<Qubit*> inputQubits, int i) {
	Qubit* qubit1 = inputQubits[0];
	Qubit* qubit2 = inputQubits[1];
	std::complex<double> result = *qubit1->fetch(i / 2) * *qubit2->fetch(i % 2);
	return make_hipDoubleComplex(result.real(), result.imag());
}


Qubit* GPUQubitFactory::generateQubit()
{
	// Allocate heap memory for Qubit values
	std::complex<double>* s0 = new std::complex<double>;
	std::complex<double>* s1 = new std::complex<double>;
	*s0 = 1.0;
	*s1 = 0.0;
	// Allocate heap memory for Qubit and store values
	Qubit* generatedQubit = new Qubit(s0, s1);
	// Push into qubit tracker for deletion
	qubits_.push_back(generatedQubit);

	return generatedQubit;
}

GPUQubitFactory::~GPUQubitFactory()
{
	for (auto qubit : qubits_) {
		delete qubit->fetch(0);
		delete qubit->fetch(1);
		delete qubit;
	}
}

Gate* GPUGateFactory::generateGate(GateRequest request)
{
	std::vector<std::vector<std::complex<double>>> gateMatrix = getGateMatrixGPU(request.getGateType());
	int gateM = gateMatrix.size();
	int gateN = gateMatrix[0].size();

	Gate* generatedGate = new Gate(gateM, gateN, gateMatrix);
	gates_.push_back(generatedGate);
	return generatedGate;
}

GPUGateFactory::~GPUGateFactory()
{
	for (auto gate : gates_) {
		delete gate;
	}
}

void GPUQuantumCircuit::loadQubitMap(std::map<std::string, std::vector<Qubit*>> qubitMap)
{
	qubitMap_ = qubitMap;
}

void GPUQuantumCircuit::loadBlock(ConcurrentBlock block)
{
	std::vector<GateRequest> gates = block.getGates();
	std::vector<Calculation> calcs;
	for (auto gate : gates) {
		std::vector<std::string> registers = gate.getRegisters();
		std::vector<int> locations = gate.getLocations();
		std::vector<Qubit*> qubitValues;
		for (int i = 0; i < registers.size(); i++) {
			qubitValues.push_back(qubitMap_[registers[i]][locations[i]]);
		}
		Gate* gateTrue = gateFactory_->generateGate(gate);
		Calculation calc = Calculation(gateTrue, qubitValues);
		calcs.push_back(calc);
	}
	calculations_.push_back(calcs);
}

std::vector<Calculation> GPUQuantumCircuit::getNextCalculation()
{
	if (calcCounter == calculations_.size() - 1) {
		done_ = true;
		return calculations_[calcCounter];
	}
	else {
		std::vector<Calculation> val = calculations_[calcCounter];
		calcCounter++;
		return val;
	}
}

std::map<std::string, std::vector<Qubit*>> GPUQuantumCircuit::returnResults()
{
	return qubitMap_;
}

bool GPUQuantumCircuit::checkComplete()
{
	if (calculations_.size() == 0) {
		return true;
	}
	return done_;
}

void GPUQuantumProcessor::loadCircuit(AbstractQuantumCircuit* circuit)
{
	circuit_ = circuit;
}

void GPUQuantumProcessor::calculate()
{
	// Generate initial arrays
	hipDoubleComplex* initialValues;
	hipDoubleComplex* beforeGate;
	hipDoubleComplex* gateValues;
	hipDoubleComplex* afterGate;
	while (!circuit_->checkComplete()) {
		std::vector<Calculation> calcBlock = circuit_->getNextCalculation();
		for (auto calc : calcBlock) {	// parallelisation next iteration
			Gate* gate = calc.getGate();
			int m = gate->getM();
			int n = gate->getN();
			int qubitN = m / 2;

			hipError_t cudaStatus;			
			// Allocate shared space
			cudaStatus = hipMalloc((void**)&initialValues, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&beforeGate, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&afterGate, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&gateValues, (m*n) * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}

			std::vector<Qubit*> qubits = calc.getQubits();

			// Generate Host side arrays for qubit values
			if (m == 2) {
				const int arraySize = 2;
				const hipDoubleComplex before[arraySize] = { convertQubitComplex(*(qubits[0]->fetch(0))), convertQubitComplex(*qubits[0]->fetch(1))};
				const hipDoubleComplex gateVal[4] = { convertQubitComplex(gate->fetchValue(0,0)), convertQubitComplex(gate->fetchValue(0,1)), convertQubitComplex(gate->fetchValue(1,0)), convertQubitComplex(gate->fetchValue(1,1)) };
				hipDoubleComplex after[arraySize] = { 0 };

				// Copy input vectors into CUDA memory
				cudaStatus = hipMemcpy(beforeGate, before, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
				cudaStatus = hipMemcpy(gateValues, gateVal, (m*n) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
				cudaStatus = hipMemcpy(afterGate, after, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}

				// Run matrix calc kernel
				ValkGPULib::matrixMul << <1, 2 >> > (afterGate, beforeGate, gateValues, 2);

				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
					goto Error;
				}

				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					goto Error;
				}

				// Copy output vector from GPU buffer to host memory.
				cudaStatus = hipMemcpy(after, afterGate, m * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
				Qubit* qubit = qubits[0];
				*qubit->fetch(0) = convertComplexQubit(after[0]);
				*qubit->fetch(1) = convertComplexQubit(after[1]);
			}
			else if (m == 4) {
				const int arraySize = 4;
				const hipDoubleComplex before[arraySize] = { tensorProduct(qubits, 0), tensorProduct(qubits, 1) , tensorProduct(qubits, 2), tensorProduct(qubits, 3) };
				const hipDoubleComplex gateVal[16] = { 
					convertQubitComplex(gate->fetchValue(0,0)), convertQubitComplex(gate->fetchValue(0,1)), convertQubitComplex(gate->fetchValue(0,2)), convertQubitComplex(gate->fetchValue(0,3)),
					convertQubitComplex(gate->fetchValue(1,0)), convertQubitComplex(gate->fetchValue(1,1)), convertQubitComplex(gate->fetchValue(1,2)), convertQubitComplex(gate->fetchValue(1,3)),
					convertQubitComplex(gate->fetchValue(2,0)), convertQubitComplex(gate->fetchValue(2,1)), convertQubitComplex(gate->fetchValue(2,2)), convertQubitComplex(gate->fetchValue(2,3)),
					convertQubitComplex(gate->fetchValue(3,0)), convertQubitComplex(gate->fetchValue(3,1)), convertQubitComplex(gate->fetchValue(3,2)), convertQubitComplex(gate->fetchValue(3,3)),
				};
				hipDoubleComplex after[arraySize] = { 0 };

				// Copy input vectors into CUDA memory
				cudaStatus = hipMemcpy(beforeGate, before, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
				cudaStatus = hipMemcpy(gateValues, gateVal, (m * n) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
				cudaStatus = hipMemcpy(afterGate, after, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}

				// Run matrix calc kernel
				ValkGPULib::matrixMul << <1, 4 >> > (afterGate, beforeGate, gateValues, 4);

				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
					goto Error;
				}

				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
					goto Error;
				}

				// Copy output vector from GPU buffer to host memory.
				cudaStatus = hipMemcpy(after, afterGate, m * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
				Qubit* qubit = qubits[0];
				*qubit->fetch(0) = convertComplexQubit(after[0]) + convertComplexQubit(after[1]);
				*qubit->fetch(1) = convertComplexQubit(after[2]) + convertComplexQubit(after[3]);
				qubit = qubits[1];
				*qubit->fetch(0) = convertComplexQubit(after[0]) + convertComplexQubit(after[2]);
				*qubit->fetch(1) = convertComplexQubit(after[1]) + convertComplexQubit(after[3]);
			}
			hipFree(initialValues);
			hipFree(beforeGate);
			hipFree(afterGate);
			hipFree(gateValues);
		}
	}
Error:
	hipFree(initialValues);
	hipFree(beforeGate);
	hipFree(afterGate);
	hipFree(gateValues);
}

std::map<std::string, std::vector<Qubit*>> GPUQuantumProcessor::qubitMapfetchQubitValues()
{
	return circuit_->returnResults();
}

void GPUDevice::loadRegister(Register registerx)
{
	if (registerx.isQuantum()) {
		QuantumRegister qReg = registerx.getQuantumRegister();
		std::string regName = qReg.getIdentifier();
		int width = qReg.getWidth();
		std::vector<Qubit*> registerQubits;
		for (int i = 0; i < width; i++) {
			registerQubits.push_back(qubitFactory->generateQubit());
		}
		registerMap.insert(std::pair<std::string, std::vector<Qubit*>>(regName, registerQubits));
	}
}

void GPUDevice::transferQubitMap()
{
	quantumCircuit->loadQubitMap(registerMap);
}

void GPUDevice::loadConcurrentBlock(ConcurrentBlock block)
{
	quantumCircuit->loadBlock(block);
}

void GPUDevice::runSimulation()
{
	quantumProcessor->loadCircuit(quantumCircuit);
	quantumProcessor->calculate();
}

void GPUDevice::run(std::vector<Register> registers, std::vector<ConcurrentBlock> blocks)
{
	for (auto reg : registers) {
		loadRegister(reg);
	}
	transferQubitMap();
	for (auto block : blocks) {
		loadConcurrentBlock(block);
	}
	runSimulation();
}

std::map<std::string, std::vector<Qubit*>> GPUDevice::revealQuantumState()
{
	return quantumProcessor->qubitMapfetchQubitValues();
}
