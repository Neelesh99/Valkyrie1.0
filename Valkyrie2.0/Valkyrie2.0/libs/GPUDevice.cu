#include "hip/hip_runtime.h"
﻿#pragma once
#include "GPUDevice.cuh"
#include "hip/hip_complex.h"
#include <cmath>
#include <stdio.h>
#include "GPUCompute.cuh"
#include "GateUtilitiesGPU.cuh"
#include <chrono>

using namespace std::complex_literals;
const double ROOT2INV = 1.0 / std::pow(2, 0.5);

/*
	GPUDevice.cu
	Description: This file defines the implementation of the functions defined
	in GPUDevice.cuh

	Defined Classes:
	GPUQubitFactory
	GPUGateFactory
	GPUQuantumCircuit
	GPUQuantumProcessor
	GPUDevice

*/

// getGateMatrix gneerates basic primitive gates (U, CX)
// uses buildU3GateGPU to construct the parameterised U gate.
std::vector<std::vector<std::complex<double>>> getGateMatrixGPU(GateRequest gate) {
	GateRequestType gateType = gate.getGateType();
	switch (gateType) {
	case I:
		return std::vector<std::vector<std::complex<double>>> { {1, 0}, { 0, 1 } };
		break;
	case h:
		return std::vector<std::vector<std::complex<double>>> { {ROOT2INV, ROOT2INV}, { ROOT2INV, -1.0 * ROOT2INV } };
		break;
	case cx:
		return std::vector<std::vector<std::complex<double>>> { {1, 0, 0, 0}, { 0, 1, 0, 0 }, { 0, 0, 0, 1 }, { 0, 0, 1, 0 } };
		break;
	case U:
		return buildU3GateGPU(gate);
		break;
	case CX:
		return std::vector<std::vector<std::complex<double>>> { {1, 0, 0, 0}, { 0, 1, 0, 0 }, { 0, 0, 0, 1 }, { 0, 0, 1, 0 } };
		break;
	}
}

// generateQubit allocates heap memory for complex number and loads it into
// a heap memory allocated Qubit and tracks the generated qubits
Qubit* GPUQubitFactory::generateQubit()
{
	// Allocate heap memory for Qubit values
	std::complex<double>* s0 = new std::complex<double>;
	std::complex<double>* s1 = new std::complex<double>;
	*s0 = 1.0;
	*s1 = 0.0;
	// Allocate heap memory for Qubit and store values
	Qubit* generatedQubit = new Qubit(s0, s1);
	// Push into qubit tracker for deletion
	qubits_.push_back(generatedQubit);

	return generatedQubit;
}

// deconstructor cleans up any heap memory allocation
GPUQubitFactory::~GPUQubitFactory()
{
	for (auto qubit : qubits_) {
		delete qubit->fetch(0);
		delete qubit->fetch(1);
		delete qubit;
	}
}

// generateQubit allocates heap memory for complex numbers and loads it into
// a heap memory allocated Gate and tracks the generated gates
Gate* GPUGateFactory::generateGate(GateRequest request)
{
	std::vector<std::vector<std::complex<double>>> gateMatrix = getGateMatrixGPU(request);
	int gateM = gateMatrix.size();
	int gateN = gateMatrix[0].size();

	Gate* generatedGate = new Gate(gateM, gateN, gateMatrix);
	gates_.push_back(generatedGate);
	return generatedGate;
}

// deconstructor cleans up any heap memory allocation
GPUGateFactory::~GPUGateFactory()
{
	for (auto gate : gates_) {
		delete gate;
	}
}

// zipSVPairs zips together identifiers and locations to generate SVPairs which can be used in
// statevector lookup
std::vector<SVPair> GPUQuantumCircuit::zipSVPairs(std::vector<std::string> names, std::vector<int> locs)
{
	std::vector<SVPair> values;
	for (int i = 0; i < names.size(); i++) {
		values.push_back(SVPair(names[i], locs[i]));
	}
	return values;
}

void GPUQuantumCircuit::loadQubitMap(std::map<std::string, std::vector<Qubit*>> qubitMap)
{
	qubitMap_ = qubitMap;
	sv_ = new StateVector(&qubitMap_);
	sv_->tensorProduct();
}

// loadBlock takes a concurrent block from the Staging module and converts it into
// a series if operable Calculation datatypes
void GPUQuantumCircuit::loadBlock(ConcurrentBlock block)
{
	std::vector<GateRequest> gates = block.getGates();
	std::vector<Calculation> calcs;
	for (auto gate : gates) {
		std::vector<std::string> registers = gate.getRegisters();
		std::vector<int> locations = gate.getLocations();
		std::vector<Qubit*> qubitValues;
		for (int i = 0; i < registers.size(); i++) {
			qubitValues.push_back(qubitMap_[registers[i]][locations[i]]);
		}
		Gate* gateTrue = gateFactory_->generateGate(gate);
		std::vector<SVPair> svPairs = zipSVPairs(registers, locations);
		Calculation calc = Calculation(gateTrue, qubitValues, svPairs);
		calcs.push_back(calc);
	}
	calculations_.push_back(calcs);
}

// getNextCalculation is used during the processing, to queue up calculations and 
// raises the done_ flag if computation is complete
std::vector<Calculation> GPUQuantumCircuit::getNextCalculation()
{
	if (calcCounter == calculations_.size() - 1) {
		done_ = true;
		return calculations_[calcCounter];
	}
	else {
		std::vector<Calculation> val = calculations_[calcCounter];
		calcCounter++;
		return val;
	}
}

// For fast computation
std::map<std::string, std::vector<Qubit*>> GPUQuantumCircuit::returnResults()
{
	return qubitMap_;
}

// For Statevector computation
StateVector* GPUQuantumCircuit::getStateVector()
{
	return sv_;
}

bool GPUQuantumCircuit::checkComplete()
{
	if (calculations_.size() == 0) {
		return true;
	}
	return done_;
}


void GPUQuantumProcessor::loadCircuit(AbstractQuantumCircuit* circuit)
{
	circuit_ = circuit;
}

// calculate method for isolated fast computation
void GPUQuantumProcessor::calculate()
{
	// Generate initial arrays
	//hipDoubleComplex* initialValues;
	hipDoubleComplex* beforeGate;
	hipDoubleComplex* gateValues;
	hipDoubleComplex* afterGate;
	while (!circuit_->checkComplete()) {
		std::vector<Calculation> calcBlock = circuit_->getNextCalculation();
		for (auto calc : calcBlock) {	// parallelisation next iteration			
			Gate* gate = calc.getGate();
			int m = gate->getM();
			int n = gate->getN();
			int qubitN = m / 2;
			hipError_t cudaStatus;			
			// Allocate shared space
			cudaStatus = hipMalloc((void**)&beforeGate, m * sizeof(hipDoubleComplex));				// Allocate GPU memory for gate arrays
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&afterGate, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&gateValues, (m*n) * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			// Uses GPUCompute.cuh functions to perform calculation
			std::vector<std::complex<double>> res = ValkGPULib::calculateGPU(beforeGate, gateValues, afterGate, calc.getGate(), calc.getQubits());
			if (res.size() == 2) {
				circuit_->getStateVector()->quickRefresh();
			}
			if (res.size() == 4) {
				circuit_->getStateVector()->modifyState(res, calc.getLocations()[0], calc.getLocations()[1]);
			}
			hipFree(beforeGate);
			hipFree(afterGate);
			hipFree(gateValues);
		}
	}
	return;
Error:
	hipFree(beforeGate);
	hipFree(afterGate);
	hipFree(gateValues);
}

// calculateWithStateVector for accurate Quantum Computer emulation, uses statevector in it's entirety
void GPUQuantumProcessor::calculateWithStateVector()
{
	// Generate initial arrays
	//hipDoubleComplex* initialValues;
	hipDoubleComplex* beforeGate;
	hipDoubleComplex* gateValues;
	hipDoubleComplex* afterGate;
	long long counter = 0;
	while (!circuit_->checkComplete()) {	// check if there are still calculations to consume
		std::vector<Calculation> calcBlock = circuit_->getNextCalculation();	// fetch calculation
		for (auto calc : calcBlock) {			
			Gate* gate = calc.getGate();
			int m = gate->getM();
			int n = gate->getN();
			int qubitN = m / 2;
			StateVector* sv = circuit_->getStateVector();						// get current state vector
			int gateDim = sv->getState().size();
			std::vector<SVPair> newOrder = calc.getNewOrder(sv->getOrder());	// use the calculation function to work out the new order of the state vector for tail procedure
			StateVector* reordered = sv->reorder(newOrder);						// fetch temporary statevector using reordered tensor product
			std::vector<std::vector<std::complex<double>>> gateValuesV = gate->getArray();
			hipError_t cudaStatus;
			// Allocate shared space
			cudaStatus = hipMalloc((void**)&beforeGate, gateDim * sizeof(hipDoubleComplex));		// Allocate GPU memory for gate arrays
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&afterGate, gateDim * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&gateValues, (m * m) * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			std::vector<std::complex<double>> res;
			res = ValkGPULib::calculateGPUSVPrime(beforeGate, gateValues, afterGate, reordered, gateValuesV, m);
			reordered->directModify(res);								// set newValues of reordered state vector
			sv->reconcile(reordered);									// reconcile temporary order for statevector for the original order
			hipFree(beforeGate);
			hipFree(afterGate);
			hipFree(gateValues);
		}
	}
	return;
Error:
	hipFree(beforeGate);
	hipFree(afterGate);
	hipFree(gateValues);
}

std::map<std::string, std::vector<Qubit*>> GPUQuantumProcessor::qubitMapfetchQubitValues()
{
	return circuit_->returnResults();
}

void GPUDevice::loadRegister(Register registerx)
{
	if (registerx.isQuantum()) {
		QuantumRegister qReg = registerx.getQuantumRegister();
		std::string regName = qReg.getIdentifier();
		int width = qReg.getWidth();
		std::vector<Qubit*> registerQubits;
		for (int i = 0; i < width; i++) {
			registerQubits.push_back(qubitFactory->generateQubit());
		}
		registerMap.insert(std::pair<std::string, std::vector<Qubit*>>(regName, registerQubits));
	}
}

void GPUDevice::transferQubitMap()
{
	quantumCircuit->loadQubitMap(registerMap);
}

void GPUDevice::loadConcurrentBlock(ConcurrentBlock block)
{
	quantumCircuit->loadBlock(block);
}

void GPUDevice::runSimulation()
{
	quantumProcessor->loadCircuit(quantumCircuit);
	quantumProcessor->calculate();
}

void GPUDevice::runSimulationSV()
{
	quantumProcessor->loadCircuit(quantumCircuit);
	quantumProcessor->calculateWithStateVector();
}

void GPUDevice::run(std::vector<Register> registers, std::vector<ConcurrentBlock> blocks)
{
	for (auto reg : registers) {
		loadRegister(reg);
	}
	transferQubitMap();
	for (auto block : blocks) {
		loadConcurrentBlock(block);
	}
	runSimulation();
}

void GPUDevice::runSV(std::vector<Register> registers, std::vector<ConcurrentBlock> blocks)
{
	for (auto reg : registers) {
		loadRegister(reg);
	}
	transferQubitMap();
	for (auto block : blocks) {
		loadConcurrentBlock(block);
	}
	runSimulationSV();
}

std::map<std::string, std::vector<Qubit*>> GPUDevice::revealQuantumState()
{
	return quantumProcessor->qubitMapfetchQubitValues();
}
