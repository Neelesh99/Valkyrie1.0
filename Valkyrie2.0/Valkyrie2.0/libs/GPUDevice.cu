#include "hip/hip_runtime.h"
﻿#pragma once
#include "GPUDevice.cuh"
#include "hip/hip_complex.h"
#include <cmath>
#include <stdio.h>
#include "GPUCompute.cuh"
#include "GateUtilitiesGPU.cuh"

using namespace std::complex_literals;
const double ROOT2INV = 1.0 / std::pow(2, 0.5);



std::vector<std::vector<std::complex<double>>> getGateMatrixGPU(GateRequest gate) {
	GateRequestType gateType = gate.getGateType();
	switch (gateType) {
	case I:
		return std::vector<std::vector<std::complex<double>>> { {1, 0}, { 0, 1 } };
		break;
	case h:
		return std::vector<std::vector<std::complex<double>>> { {ROOT2INV, ROOT2INV}, { ROOT2INV, -1.0 * ROOT2INV } };
		break;
	case cx:
		return std::vector<std::vector<std::complex<double>>> { {1, 0, 0, 0}, { 0, 1, 0, 0 }, { 0, 0, 0, 1 }, { 0, 0, 1, 0 } };
		break;
	case U:
		return buildU3GateGPU(gate);
		break;
	case CX:
		return std::vector<std::vector<std::complex<double>>> { {1, 0, 0, 0}, { 0, 1, 0, 0 }, { 0, 0, 0, 1 }, { 0, 0, 1, 0 } };
		break;
	}
}


Qubit* GPUQubitFactory::generateQubit()
{
	// Allocate heap memory for Qubit values
	std::complex<double>* s0 = new std::complex<double>;
	std::complex<double>* s1 = new std::complex<double>;
	*s0 = 1.0;
	*s1 = 0.0;
	// Allocate heap memory for Qubit and store values
	Qubit* generatedQubit = new Qubit(s0, s1);
	// Push into qubit tracker for deletion
	qubits_.push_back(generatedQubit);

	return generatedQubit;
}

GPUQubitFactory::~GPUQubitFactory()
{
	for (auto qubit : qubits_) {
		delete qubit->fetch(0);
		delete qubit->fetch(1);
		delete qubit;
	}
}

Gate* GPUGateFactory::generateGate(GateRequest request)
{
	std::vector<std::vector<std::complex<double>>> gateMatrix = getGateMatrixGPU(request);
	int gateM = gateMatrix.size();
	int gateN = gateMatrix[0].size();

	Gate* generatedGate = new Gate(gateM, gateN, gateMatrix);
	gates_.push_back(generatedGate);
	return generatedGate;
}

GPUGateFactory::~GPUGateFactory()
{
	for (auto gate : gates_) {
		delete gate;
	}
}

std::vector<SVPair> GPUQuantumCircuit::zipSVPairs(std::vector<std::string> names, std::vector<int> locs)
{
	std::vector<SVPair> values;
	for (int i = 0; i < names.size(); i++) {
		values.push_back(SVPair(names[i], locs[i]));
	}
	return values;
}

void GPUQuantumCircuit::loadQubitMap(std::map<std::string, std::vector<Qubit*>> qubitMap)
{
	qubitMap_ = qubitMap;
	sv_ = new StateVector(&qubitMap_);
	sv_->tensorProduct();
}

void GPUQuantumCircuit::loadBlock(ConcurrentBlock block)
{
	std::vector<GateRequest> gates = block.getGates();
	std::vector<Calculation> calcs;
	for (auto gate : gates) {
		std::vector<std::string> registers = gate.getRegisters();
		std::vector<int> locations = gate.getLocations();
		std::vector<Qubit*> qubitValues;
		for (int i = 0; i < registers.size(); i++) {
			qubitValues.push_back(qubitMap_[registers[i]][locations[i]]);
		}
		Gate* gateTrue = gateFactory_->generateGate(gate);
		std::vector<SVPair> svPairs = zipSVPairs(registers, locations);
		Calculation calc = Calculation(gateTrue, qubitValues, svPairs);
		calcs.push_back(calc);
	}
	calculations_.push_back(calcs);
}

std::vector<Calculation> GPUQuantumCircuit::getNextCalculation()
{
	if (calcCounter == calculations_.size() - 1) {
		done_ = true;
		return calculations_[calcCounter];
	}
	else {
		std::vector<Calculation> val = calculations_[calcCounter];
		calcCounter++;
		return val;
	}
}

std::map<std::string, std::vector<Qubit*>> GPUQuantumCircuit::returnResults()
{
	return qubitMap_;
}

StateVector* GPUQuantumCircuit::getStateVector()
{
	return sv_;
}

bool GPUQuantumCircuit::checkComplete()
{
	if (calculations_.size() == 0) {
		return true;
	}
	return done_;
}

void GPUQuantumProcessor::loadCircuit(AbstractQuantumCircuit* circuit)
{
	circuit_ = circuit;
}

void GPUQuantumProcessor::calculate()
{
	// Generate initial arrays
	//hipDoubleComplex* initialValues;
	hipDoubleComplex* beforeGate;
	hipDoubleComplex* gateValues;
	hipDoubleComplex* afterGate;
	while (!circuit_->checkComplete()) {
		std::vector<Calculation> calcBlock = circuit_->getNextCalculation();
		for (auto calc : calcBlock) {	// parallelisation next iteration			
			Gate* gate = calc.getGate();
			int m = gate->getM();
			int n = gate->getN();
			int qubitN = m / 2;
			hipError_t cudaStatus;			
			// Allocate shared space
			cudaStatus = hipMalloc((void**)&beforeGate, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&afterGate, m * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			cudaStatus = hipMalloc((void**)&gateValues, (m*n) * sizeof(hipDoubleComplex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				goto Error;
			}
			std::vector<std::complex<double>> res = ValkGPULib::calculateGPU(beforeGate, gateValues, afterGate, calc.getGate(), calc.getQubits());
			if (res.size() == 2) {
				circuit_->getStateVector()->quickRefresh();
			}
			if (res.size() == 4) {
				circuit_->getStateVector()->modifyState(res, calc.getLocations()[0], calc.getLocations()[1]);
			}
			hipFree(beforeGate);
			hipFree(afterGate);
			hipFree(gateValues);
		}
	}
	return;
Error:
	hipFree(beforeGate);
	hipFree(afterGate);
	hipFree(gateValues);
}

std::map<std::string, std::vector<Qubit*>> GPUQuantumProcessor::qubitMapfetchQubitValues()
{
	return circuit_->returnResults();
}

void GPUDevice::loadRegister(Register registerx)
{
	if (registerx.isQuantum()) {
		QuantumRegister qReg = registerx.getQuantumRegister();
		std::string regName = qReg.getIdentifier();
		int width = qReg.getWidth();
		std::vector<Qubit*> registerQubits;
		for (int i = 0; i < width; i++) {
			registerQubits.push_back(qubitFactory->generateQubit());
		}
		registerMap.insert(std::pair<std::string, std::vector<Qubit*>>(regName, registerQubits));
	}
}

void GPUDevice::transferQubitMap()
{
	quantumCircuit->loadQubitMap(registerMap);
}

void GPUDevice::loadConcurrentBlock(ConcurrentBlock block)
{
	quantumCircuit->loadBlock(block);
}

void GPUDevice::runSimulation()
{
	quantumProcessor->loadCircuit(quantumCircuit);
	quantumProcessor->calculate();
}

void GPUDevice::run(std::vector<Register> registers, std::vector<ConcurrentBlock> blocks)
{
	for (auto reg : registers) {
		loadRegister(reg);
	}
	transferQubitMap();
	for (auto block : blocks) {
		loadConcurrentBlock(block);
	}
	runSimulation();
}

std::map<std::string, std::vector<Qubit*>> GPUDevice::revealQuantumState()
{
	return quantumProcessor->qubitMapfetchQubitValues();
}
