﻿#include "GateFactory.cuh"

// CPU Gate Factory
Gate* CPUGateFactory::createNewCNOTGate() {
	int* dimensions = new int[2]{ 4,4 };
	Gate* CNOTCandidate = new CPUGate();
	CNOTCandidate->setDimensions(dimensions);
	valk::ComplexNumber* values = new valk::ComplexNumber[16]{
		1, 0, 0, 0,
		0, 1, 0, 0,
		0, 0, 0, 1,
		0, 0, 1, 0
	};
	CNOTCandidate->setValues(values);
	return CNOTCandidate;
}

Gate* CPUGateFactory::createNewUGate(valk::ComplexNumber* values) {
	int* dimensions = new int[2]{ 2,2 };
	Gate* UGateCandidate = new CPUGate();
	UGateCandidate->setDimensions(dimensions);
	UGateCandidate->setValues(values);
	return UGateCandidate;
}

void CPUGateFactory::appendToGates(Gate* val) {
	if (empty) {
		gates_emitted_[0] = val;
		empty = false;
	}
	else {
		Gate** newGateList = new Gate * [length_ + 1];
		memcpy(newGateList, gates_emitted_, length_);
		Gate** oldTracker = gates_emitted_;
		newGateList[length_] = val;
		length_++;
		gates_emitted_ = newGateList;
		delete oldTracker;
	}
}

Gate* CPUGateFactory::createNewGate(valk::availableGates gate_types) {
	Gate* returnGate = createNewCNOTGate();
	switch (gate_types) {
	case valk::availableGates::CNOT:		
		appendToGates(returnGate);
		return returnGate;
		break;
	case valk::availableGates::U:
		throw std::invalid_argument("Expected list of initialised values for U gate");
		break;
	default:
		throw std::invalid_argument("Gate not supported");
	}
}

Gate* CPUGateFactory::createNewGate(valk::availableGates gate_types, valk::ComplexNumber* values) {
	Gate* returnGate = createNewUGate(values);
	switch (gate_types) {
	case valk::availableGates::CNOT:
		throw std::invalid_argument("Expected no initialised values for CNOT gate");
		break;
	case valk::availableGates::U:		
		appendToGates(returnGate);
		return returnGate;
		break;
	default:
		throw std::invalid_argument("Gate not supported");
	}
}

CPUGateFactory::~CPUGateFactory() {
	if (empty) {
		delete gates_emitted_;
		return;
	}
	// Delete gate data
	for (int i = 0; i < length_; i++) {
		Gate* resolved_ptr = gates_emitted_[i];
		delete resolved_ptr->getDimensions();
		delete resolved_ptr->getValues();
		delete resolved_ptr;
	}
	delete gates_emitted_;
}

//GPU Gate Factory
Gate* GPUGateFactory::createNewCNOTGate() {
	int* dimensions;
	hipMallocManaged(&dimensions, 2 * sizeof(int));
	dimensions[0] = 4;
	dimensions[1] = 4;
	Gate* CNOTCandidate;
	hipMallocManaged(&CNOTCandidate, sizeof(GPUGate));
	CNOTCandidate = new(CNOTCandidate) GPUGate();
	CNOTCandidate->setDimensions(dimensions);
	valk::ComplexNumber* values;
	hipMallocManaged(&values, 16 * sizeof(valk::ComplexNumber));
	valk::ComplexNumber values_const[] = {
		1, 0, 0, 0,
		0, 1, 0, 0,
		0, 0, 0, 1,
		0, 0, 1, 0
	};
	for (int i = 0; i < 16; i++) {
		values[i] = values_const[i];
	}
	CNOTCandidate->setValues(values);
	return CNOTCandidate;
}

Gate* GPUGateFactory::createNewUGate(valk::ComplexNumber* values) {
	int* dimensions;
	hipMallocManaged(&dimensions, 2 * sizeof(int));
	dimensions[0] = 2;
	dimensions[1] = 2;
	Gate* UGateCandidate;
	hipMallocManaged(&UGateCandidate, sizeof(GPUGate));
	UGateCandidate = new(UGateCandidate) GPUGate();
	UGateCandidate->setDimensions(dimensions);
	UGateCandidate->setValues(values);
	return UGateCandidate;
}

void GPUGateFactory::appendToGates(Gate* val) {
	if (empty) {
		gates_emitted_[0] = val;
		empty = false;
	}
	else {
		Gate** newGateList;
		hipMallocManaged(newGateList, (length_ + 1) * sizeof(Gate*));
		hipMemcpy(newGateList, gates_emitted_, length_, hipMemcpyDefault);
		Gate** oldTracker = gates_emitted_;
		newGateList[length_] = val;
		length_++;
		gates_emitted_ = newGateList;
		hipFree(oldTracker);
	}
}

Gate* GPUGateFactory::createNewGate(valk::availableGates gate_types) {
	Gate* returnGate = createNewCNOTGate();
	switch (gate_types) {
	case valk::availableGates::CNOT:		
		appendToGates(returnGate);
		return returnGate;
		break;
	case valk::availableGates::U:
		throw std::invalid_argument("Expected list of initialised values for U gate");
		break;
	default:
		throw std::invalid_argument("Gate not supported");
	}
}

Gate* GPUGateFactory::createNewGate(valk::availableGates gate_types, valk::ComplexNumber* values) {
	Gate* returnGate = createNewUGate(values);
	switch (gate_types) {
	case valk::availableGates::CNOT:
		throw std::invalid_argument("Expected no initialised values for CNOT gate");
		break;
	case valk::availableGates::U:		
		appendToGates(returnGate);
		return returnGate;
		break;
	default:
		throw std::invalid_argument("Gate not supported");
	}
}

GPUGateFactory::~GPUGateFactory() {
	if (empty) {
		hipFree(gates_emitted_);
		return;
	}
	// Delete gate data
	for (int i = 0; i < length_; i++) {
		Gate* resolved_ptr = gates_emitted_[i];
		hipFree(resolved_ptr->getDimensions());
		hipFree(resolved_ptr->getValues());
		hipFree(resolved_ptr);
	}
	hipFree(gates_emitted_);
}


