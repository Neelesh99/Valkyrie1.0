﻿#include "QubitFactory.cuh"

Qubit* CPUQubitFactory::generateBlankQubit() {
	Qubit* quPointer;
	quPointer = new Qubit();
	if (empty) {
		qubitTracker_[0] = quPointer;
		empty = false;
	}
	else {
		Qubit** newtracker = new Qubit * [trackerSize + 1];
		memcpy(newtracker, qubitTracker_, trackerSize);
		Qubit** oldtracker = qubitTracker_;
		newtracker[trackerSize] = quPointer;
		trackerSize++;
		qubitTracker_ = newtracker;
		delete oldtracker;
	}
	int* dims = new int[2] {2, 1};
	quPointer->setQubitDimensions(dims);
	valk::ComplexNumber* values = new valk::ComplexNumber[2]{ 0,0 };
	default0_ ? values[0] = 1 : values[1] = 1;
	quPointer->safeLoadValues(values);
	return quPointer;
}

CPUQubitFactory::~CPUQubitFactory() {
	if (empty) {
		delete qubitTracker_;
	}
	else {
		for (int i = 0; i < trackerSize; i++) {
			Qubit* resolvedQubit = qubitTracker_[i];
			delete resolvedQubit->getQubitValues();
			delete resolvedQubit->getQubitDimensions();
			delete resolvedQubit;
		}
		delete qubitTracker_;
	}
}

Qubit* GPUQubitFactory::generateBlankQubit() {
	Qubit* quPointer;
	hipMallocManaged(&quPointer, sizeof(Qubit));
	if (empty) {
		qubitTracker_[0] = quPointer;
		empty = false;
	}
	else {
		Qubit** newtracker;
		hipMallocManaged(&newtracker, (trackerSize+1)*sizeof(Qubit*));
		hipMemcpy(newtracker, qubitTracker_, trackerSize, hipMemcpyDefault);
		Qubit** oldtracker =  qubitTracker_;
		newtracker[trackerSize] = quPointer;
		trackerSize++;
		qubitTracker_ = newtracker;
		hipFree(oldtracker);
	}
	int* dims;
	hipMallocManaged(&dims, 2 * sizeof(int));
	dims[0] = 2;
	dims[1] = 1;
	quPointer->setQubitDimensions(dims);
	valk::ComplexNumber* values;
	hipMallocManaged(&values, 2 * sizeof(valk::ComplexNumber));
	values[0] = 0;
	values[1] = 0;
	default0_ ? values[0] = 1 : values[1] = 1;
	quPointer->safeLoadValues(values);
	return quPointer;
}

GPUQubitFactory::~GPUQubitFactory() {
	if (empty) {
		hipFree(qubitTracker_);
	}
	else {
		for (int i = 0; i < trackerSize; i++) {
			Qubit* resolvedQubit = qubitTracker_[i];
			hipFree(resolvedQubit->getQubitValues());
			hipFree(resolvedQubit->getQubitDimensions());
			hipFree(resolvedQubit);
		}
		hipFree(qubitTracker_);
	}
}

